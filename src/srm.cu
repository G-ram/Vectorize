#include "hip/hip_runtime.h"
#include "srm.cuh"

#define Q 32

struct edge_functor(){
	region currentRegion;
	edge_functor(){}
	__host__ __device__
	edge operator()(const frame& frame, int idx){
		return ;
	}
}
struct sort_edges{
	__host__ __device__
	bool operator()(const edge& e1, const edge& e2) {
		return e1.mag < e2.mag;
	}
}
regions SRM(thrust::device_vector<frame> frames){
	int i;
	thrust::host_vector<thrust::raw_pointer<edge>> hEdges(frames.size());
	for(i = 0; i < frames.size(); i++){
		hEdges = thrust::raw_pointer_cast(new thrust::device_vector<edge>(2*frames[0].w*frames[0]).data();
	}
	thrust::device_vector<thrust::raw_pointer<edge>> dEdges = hEdges;
	thrust::transform(frames.begin(),frames.end(),dEdges.begin,dEdges.end(),edge_functor);
	
	for(i = 0; i < frames.size(); i++){
		thrust.sort(dEdges.begin(),dEdges.end(),
	}
}
/*__global__ void srm1(region* regions, edge* pairs, int w, int numOfPixels){
	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	int i = w*y+x;
	if(i >= numOfPixels){return;}
	pixel currentPixel = regions[i].rootPixel;
	pixel eastPixel;
	pixel southPixel;
	if(i+1 < numOfPixels){
		eastPixel = regions[i+1].rootPixel;
		int magEast = max3(abs(currentPixel.r-eastPixel.r),
					abs(currentPixel.g-eastPixel.g),
					abs(currentPixel.b-eastPixel.b));
		edge tempEdge = {regions[i],regions[i+1],magEast};
		pairs[2*i] =  tempEdge;
	}
	if(i+w < numOfPixels){
		southPixel = regions[i+w].rootPixel;
		int magSouth = max3(abs(currentPixel.r-southPixel.r),
					abs(currentPixel.g-southPixel.g),
					abs(currentPixel.b-southPixel.b));
		edge tempSouth = {regions[i],regions[i+w],magSouth};
		pairs[2*i+1] = tempSouth;
	}
	__syncthreads();
	int j = 0;
	i *= 2;
	//Some form of bubble sort
	for(j = 0;j < numOfPixels;j++){
		if(j%2 == 0){
			if(pairs[i].mag > pairs[i+1].mag){
				edge tempEdge = pairs[i];
				pairs[i] = pairs[i+1];
				pairs[i+1] = tempEdge;
			}
		}else if(i+1 < 2*numOfPixels-1){
			if(pairs[i+1].mag > pairs[i+2].mag){
				edge tempEdge = pairs[i+1];
				pairs[i] = tempEdge;
			}
		}
		__syncthreads();
	}
}

bool mergeTest(region regionA, region regionB,region* regions,int size){
	float lowerCaseDelta = 1/(6*(float)(size*size));
	int numRegsWithA = 0;
	int numRegsWithB = 0;
	int i;
	for(i = 0; i < size; i++){
		if(regions[i].numOfPixels == regionA.numOfPixels){numRegsWithA++;}
		if(regions[i].numOfPixels == regionB.numOfPixels){numRegsWithB++;}
	}
	float maxAvgColorA = max3(regionA.avgR,regionA.avgG,regionA.avgB);
	float maxAvgColorB = max3(regionB.avgR,regionB.avgG,regionB.avgB);
	float bA = (1/(Q*regionA.numOfPixels*2))*log(numRegsWithA/lowerCaseDelta);
	float bB = (1/(Q*regionB.numOfPixels*2))*log(numRegsWithB/lowerCaseDelta);
	return fabs(maxAvgColorA-maxAvgColorB) <= bA*bA+bB*bB;
}

void srm2(region* regions, edge* pairs,int numOfPixels){
	int j;
	for(j = 0; j < numOfPixels*2;j++){
		region r1 = pairs[j].r1;
		region r2 = pairs[j].r2;
		if(mergeTest(r1,r2,regions,numOfPixels) == true){
			if(r1.numOfPixels >= r2.numOfPixels){
				pairs[j].r1.child = &(pairs[j].r2);
				pairs[j].r2.head = false;	
			}else{
				pairs[j].r2.child = &(pairs[j].r1);
				pairs[j].r1.head = false;
			}
			int totalPixels = r1.numOfPixels + r2.numOfPixels;
			pairs[j].r1.avgR = (r1.numOfPixels/totalPixels)*r1.avgR+(r2.numOfPixels/totalPixels)*r2.avgR;
			pairs[j].r1.avgG = (r1.numOfPixels/totalPixels)*r1.avgG+(r2.numOfPixels/totalPixels)*r2.avgG;
			pairs[j].r1.avgB = (r1.numOfPixels/totalPixels)*r1.avgB+(r2.numOfPixels/totalPixels)*r2.avgB;
			pairs[j].r2.avgR = (r1.numOfPixels/totalPixels)*r1.avgR+(r2.numOfPixels/totalPixels)*r2.avgR;
			pairs[j].r2.avgG = (r1.numOfPixels/totalPixels)*r1.avgG+(r2.numOfPixels/totalPixels)*r2.avgG;
			pairs[j].r1.avgB = (r1.numOfPixels/totalPixels)*r1.avgB+(r2.numOfPixels/totalPixels)*r2.avgB;
			pairs[j].r1.numOfPixels += pairs[j].r2.numOfPixels;
			pairs[j].r2.numOfPixels +=  pairs[j].r1.numOfPixels;
		}	
	}
}*/
