#include "hip/hip_runtime.h"
#include "srm.cuh"

#define Q 32

struct gen_regions: public thrust::unary_function<int,region>{
	frame *currentFrame;
	gen_regions(frame *_currentFrame){
		currentFrame = _currentFrame;
	}
	__device__ region operator()(const int& idx){
		region tempRegion;
		tempRegion.id = idx;
		tempRegion.numOfPixels = 1;
		tempRegion.avgR = (int)(*currentFrame).pixelData[idx].r;
		tempRegion.avgG = (int)(*currentFrame).pixelData[idx].g;
		tempRegion.avgG = (int)(*currentFrame).pixelData[idx].b;
		return tempRegion;
	}
};

struct gen_edges: public thrust::unary_function<int,edge>{
	frame *currentFrame;
	region *regions;
	unsigned int w;
	unsigned int h;
	unsigned int inc;
	gen_edges(frame *_currentFrame, region *_regions, unsigned int _w, unsigned int _h, unsigned int _inc){
		currentFrame = _currentFrame;
		regions = _regions;
		w = _w;
		h = _h;
		inc = _inc;
	}
	__device__ unsigned int max3(int x, int y, int z){
		unsigned int max = x >= y ? x : y;
		return max >= z ? max : z;
	}
	__device__ edge operator()(const int& idx){
		edge tempEdge;
		if(idx + inc < w*h){
			region r1 = regions[idx];
			region r2 = regions[idx+inc];
			tempEdge.r1 = idx;
			tempEdge.r2 = idx+inc;
			tempEdge.mag = max3(abs(r1.avgR-r2.avgR),abs(r1.avgG-r2.avgG),abs(r1.avgB-r2.avgB));
		}else{
			tempEdge.r1 = 0;
			tempEdge.r2 = 0;
			tempEdge.mag = -1.0;
		}

		return tempEdge;
	}
};

struct sort_edges{
	__device__ bool operator()(const edge& x, const edge& y){
		return x.mag < y.mag;
  	}
};

struct gen_labels: public thrust::unary_function<region,unsigned int>{
	gen_labels(){}
	__device__ unsigned int operator()(const region& currentRegion){
		return currentRegion.id;
	}
};

float max3(float x, float y, float z){
	float max = x >= y ? x : y;
	return max >= z ? max : z;
}

bool mergeTest(edge currentEdge,thrust::host_vector<region> regions, std::vector<unsigned int> sizes, float delta){
	region r1 = regions[currentEdge.r1];
	region r2 = regions[currentEdge.r2];
	unsigned int numRegionsWithPixels1 = sizes[r1.numOfPixels];
	unsigned int numRegionsWithPixels2 = sizes[r2.numOfPixels];
	float colorDiff = max3(fabs(r1.avgR-r2.avgR),fabs(r1.avgG-r2.avgG),fabs(r1.avgB-r2.avgB));
	float bR1 = (1/(2*Q*r1.numOfPixels))*log(numRegionsWithPixels1/delta);
	float bR2 = (1/(2*Q*r1.numOfPixels))*log(numRegionsWithPixels2/delta);
	return colorDiff <= sqrt(bR1+bR2);
}

thrust::device_vector<unsigned int> SRM(frame* frames, int numOfFrames, int w, int h){
	int size = w*h;
	float delta = 1/(6*size*size);
	thrust::device_vector<region> regions(size);
	thrust::device_vector<unsigned int> idxs(size);
	thrust::device_vector<edge> edges(2*size);
	thrust::device_vector<unsigned int> labels(numOfFrames*size);
	thrust::host_vector<region> hRegions(size);
	thrust::host_vector<edge> hEdges(2*size);
	std::vector<unsigned int> sizes(size+1);
	sizes[size] = size;
	thrust::sequence(idxs.begin(),idxs.end());
	int i,j,k;
	i = j = k = 0;
	for(j = 0; j < numOfFrames; j++){
		thrust::transform(idxs.begin(),idxs.end(),regions.begin(),gen_regions(frames+j));
		thrust::transform(idxs.begin(),idxs.end(),edges.begin(),gen_edges(frames+j,thrust::raw_pointer_cast(regions.data()),w,h,1));
		thrust::transform(idxs.begin(),idxs.end(),edges.begin()+size,gen_edges(frames+j,thrust::raw_pointer_cast(regions.data()),w,h,w));
		thrust::sort(edges.begin(),edges.end(),sort_edges());
		hRegions = regions;
		hEdges = edges;
		region r1, r2;
		unsigned int totalPixels;
		for(thrust::host_vector<edge>::iterator it = hEdges.begin(); it != hEdges.end(); it++){
			i++;
			if(mergeTest(hEdges[i],hRegions,sizes,delta)){ //merge O(n)
				r1 = hRegions[hEdges[i].r1];
				r2 = hRegions[hEdges[i].r2];
				totalPixels = r1.numOfPixels+r2.numOfPixels;
				sizes[totalPixels] += 1;
				sizes[r1.numOfPixels] -= 1;
				sizes[r2.numOfPixels] -= 1;
				r2.avgR = (r1.numOfPixels/totalPixels)*r1.avgR+(r2.numOfPixels/totalPixels)*r2.avgR;
				r2.avgG = (r1.numOfPixels/totalPixels)*r1.avgG+(r2.numOfPixels/totalPixels)*r2.avgB;
				r2.avgB = (r1.numOfPixels/totalPixels)*r1.avgB+(r2.numOfPixels/totalPixels)*r2.avgG;
				r2.numOfPixels = totalPixels;
				for(thrust::host_vector<region>::iterator ik = hRegions.begin(); ik != hRegions.end(); ik++){
					k++;
					if(hRegions[k].id == r1.id || hRegions[k].id == r2.id){
						hRegions[k] = r2;
					}
				}
			}
		}
		regions = hRegions;
		thrust::transform(regions.begin(),regions.end(),labels.begin()+j*size,gen_labels());
		thrust::transform(idxs.begin(),idxs.end(),regions.begin(),gen_regions(frames+j));
	}
	return labels;
}


