#include "hip/hip_runtime.h"
#include "boundary.cuh"

#define VALUE_AT(x, y)\
    segmentedImages[(y - 1) * imageWidth + (x - 1)]

#define EDGES_VALUE_AT(x, y)\
    edges[(y - 1) * imageWidth * 2 + (x - 1)]

static const int BLOCK_SIZE = 16;

void throw_on_cuda_error(const char *file, int line)
{
	hipError_t lastError = hipGetLastError();

  	if(lastError != hipSuccess) {
		std::cout << file << "(" << line << ")" << " - " <<
			hipGetErrorString(lastError) << "\n";
    	exit(1);
  	}
}

__device__ bool rangeCheck(int x, int y, int height, int width)
{
    if (x <= 0 || x > width) {
        return false;
    }

    if (y <= 0 || y > width) {
        return false;
    }

    return true;
}

__global__ void subpixelKernel(bool *edges, unsigned int *segmentedImages,
    int imageHeight, int imageWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= imageWidth * 2) {
        return;
    }

    if (y >= imageHeight * 2) {
        return;
    }

    x++;
    y++;

    // x is odd and y is even
    if (x % 2 != 0 && y % 2 == 0) {
        if (
            rangeCheck((x + 1) / 2, y / 2, imageHeight, imageWidth) &&
            rangeCheck((x - 1) / 2, y / 2, imageHeight, imageWidth) &&
            VALUE_AT((x + 1) / 2, y / 2) !=
            VALUE_AT((x - 1) / 2, y / 2)
        ) {
            EDGES_VALUE_AT(x, y) = true;
        }
    }

    // x is even and y is odd
    if (x % 2 == 0 && y % 2 != 0) {
        if (
            rangeCheck(x / 2, (y + 1) / 2, imageHeight, imageWidth) &&
            rangeCheck(x / 2, (y - 1) / 2, imageHeight, imageWidth) &&
            VALUE_AT(x / 2, (y + 1) / 2) !=
            VALUE_AT(x / 2, (y - 1) / 2)
        ) {
            EDGES_VALUE_AT(x, y) = true;
        }
    }
}

__global__ void gapKernel(bool *edges, unsigned int *segmentedImages,
    int imageHeight, int imageWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= imageWidth * 2) {
        return;
    }

    if (y >= imageHeight * 2) {
        return;
    }

    x++;
    y++;

    if (
        rangeCheck(x + 1, y, imageHeight * 2, imageWidth * 2) &&
        rangeCheck(x - 1, y, imageHeight * 2, imageWidth * 2) &&
        EDGES_VALUE_AT(x + 1, y) &&
        EDGES_VALUE_AT(x - 1, y)
    ) {
        EDGES_VALUE_AT(x, y) = true;
    }

    if (
        rangeCheck(x, y + 1, imageHeight * 2, imageWidth * 2) &&
        rangeCheck(x, y - 1, imageHeight * 2, imageWidth * 2) &&
        EDGES_VALUE_AT(x, y + 1) &&
        EDGES_VALUE_AT(x, y - 1)
    ) {
        EDGES_VALUE_AT(x, y) = true;
    }
}

__global__ void junctionKernel(bool *edges, unsigned int *segmentedImages,
    int imageHeight, int imageWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= imageWidth) {
        return;
    }

    if (y >= imageHeight) {
        return;
    }

    x++;
    y++;

    int neighborCount = 0;

    if (rangeCheck(x + 1, y, imageHeight * 2, imageWidth * 2) &&
        EDGES_VALUE_AT(x + 1, y) == true) {
        neighborCount++;
    }
    if (rangeCheck(x - 1, y, imageHeight * 2, imageWidth * 2) &&
        EDGES_VALUE_AT(x - 1, y) == true) {
        neighborCount++;
    }
    if (rangeCheck(x, y + 1, imageHeight * 2, imageWidth * 2) &&
        EDGES_VALUE_AT(x, y + 1) == true) {
        neighborCount++;
    }
    if (rangeCheck(x, y - 1, imageHeight * 2, imageWidth * 2) &&
        EDGES_VALUE_AT(x, y - 1) == true) {
        neighborCount++;
    }

    if (neighborCount > 2) {
        EDGES_VALUE_AT(x, y) = true;
    }
}

thrust::device_vector<bool>
genSubpixelEdges(thrust::device_vector<unsigned int> segmentedImages,
    int imageHeight, int imageWidth)
{
    int imageSize = imageHeight * imageWidth;
    thrust::device_vector<bool> edges(imageSize * 4);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((imageWidth * 2 + BLOCK_SIZE - 1) / BLOCK_SIZE,
      (imageHeight * 2 + BLOCK_SIZE - 1) / BLOCK_SIZE);


    subpixelKernel<<<numBlocks, threadsPerBlock>>>(
        thrust::raw_pointer_cast(edges.data()),
        thrust::raw_pointer_cast(segmentedImages.data()),
        imageHeight,
        imageWidth
    );
    throw_on_cuda_error(__FILE__, __LINE__);

    gapKernel<<<numBlocks, threadsPerBlock>>>(
        thrust::raw_pointer_cast(edges.data()),
        thrust::raw_pointer_cast(segmentedImages.data()),
        imageHeight,
        imageWidth
    );
    throw_on_cuda_error(__FILE__, __LINE__);

    // junctionKernel<<<numBlocks, threadsPerBlock>>>(
    //     thrust::raw_pointer_cast(edges.data()),
    //     thrust::raw_pointer_cast(segmentedImages.data()),
    //     imageHeight,
    //     imageWidth
    // );
    throw_on_cuda_error(__FILE__, __LINE__);

    return edges;
}

int
testBoundary()
{
    int height = 5;
    int width = 5;

    unsigned int test[] = {
        1, 1, 1, 2, 2,
        1, 1, 3, 2, 2,
        1, 3, 3, 4, 2,
        3, 3, 4, 4, 4,
        3, 3, 4, 4, 4
    };
    thrust::device_vector<unsigned int> segmentedImages(test,
        test + height * width);

    thrust::host_vector<bool> edges =
        genSubpixelEdges(segmentedImages, height, width);

    for (int i = 0; i < edges.size(); i++) {
        std::cout << edges[i] << " ";
        if ((i + 1) % (width * 2) == 0) {
            std::cout << "\n";
        }
    }

    return 0;
}
