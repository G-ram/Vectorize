#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "video.h"
#include "struct.h"
// #include "srm.cuh"

static const int FRAME_BATCH_SIZE = 100;

compactVideoRead
copyPictureDataToDev(frame *frames, pixel *pixels, unsigned int framesRead,
	unsigned int imageSize)
{
	int i;
	compactVideoRead devRead;
	CUDA_CALL_SETUP;
	pixel *devPixels;
	frame *devFrames;

	// Prints out the first pixel of each frame
	// cout << *pixels << "\n";

	CUDA_CALL(hipMalloc((void**) &devPixels,
		sizeof(pixel) * framesRead * imageSize),
		"hipMalloc failed - devPixels");

	CUDA_CALL(hipMemcpy(devPixels, pixels,
		sizeof(pixel) * framesRead * imageSize, hipMemcpyHostToDevice),
		"hipMemcpy failed - devPixels");

	// Setup pointers to dev pixel data in frames
	// before copying to frames to dev
	for (i = 0; i < framesRead; i++) {
		frames[i].pixelData = devPixels + i * imageSize;
	}

	CUDA_CALL(hipMalloc((void**) &devFrames,
		sizeof(frame) * framesRead),
		"hipMalloc failed - devFrames");

	CUDA_CALL(hipMemcpy(devFrames, frames,
		sizeof(frame) * framesRead, hipMemcpyHostToDevice),
		"hipMemcpy failed - devFrames");

	devRead.pixels = devPixels;
	devRead.frames = devFrames;

	return devRead;

Error:
	hipFree(devPixels);
	hipFree(devFrames);

	exit(1);
}

int main(int argc, char** argv ) {
	if(argc != 2) {
		printf("usage: <Video path>\n");
		return -1;
	}

	Video video = Video(argv[1]);

	cout << "Reading from video file: " << argv[1] << "\n";
	while(video.hasNext()) {
		unsigned int imageSize = video.getHeight() * video.getWidth();
		compactVideoRead read = video.readNFrames(FRAME_BATCH_SIZE);
		assert(read.framesRead > 0);
		cout << read.framesRead << " frames read \n";

		compactVideoRead devRead =
			copyPictureDataToDev(read.frames, read.pixels, read.framesRead,
				imageSize);

		assert(devRead.frames != NULL);
	}
	return 0;
}
