#include "hip/hip_runtime.h"
#include <stdio.h>
#include "video.h"
#include "struct.h"
#include "srm.cuh"
__global__ void setup(uchar* rawPixels, region* regions, int w, int h){
	int x = blockDim.x*blockIdx.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	int i = 3*w*y+3*x;
	int ip = w*y+x;
	if(x >= w || y >= h){return;}
	int b = (int)rawPixels[i];
	int g = (int)rawPixels[i+1];
	int r = (int)rawPixels[i+2];
	pixel tempPixel = {i,r,g,b};
	region tempRegion = {true,NULL,tempPixel,1,r,g,b};
	regions[ip] = tempRegion;
}
int main(int argc, char** argv ){
	if(argc != 2){
		printf("usage: <Image_Path>\n");
		return -1;
	}
	Video video = Video(argv[1]);
	while(video.next() != NULL){

	}
	return 0;
}

